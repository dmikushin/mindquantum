#include "hip/hip_runtime.h"
//   Copyright 2021 <Huawei Technologies Co., Ltd>
//
//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at
//
//       http://www.apache.org/licenses/LICENSE-2.0
//
//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.

#include "debug_info.hpp"
#include "gmem.h"
#include "nvidia/check.h"

#include <hip/hip_runtime.h>

#include <dlfcn.h>

#include <cstdio>
#include <fstream>
#include <iterator>
#include <map>
#include <string>
#include <vector>

__device__ double2 m_const[32 * 32];  // kernel5 (1 << N) * (1 << N)

extern "C" void* load_m_const(const void* data, size_t size)
{
    if (size > sizeof(m_const)) {
        fprintf(stderr, "load_m_const() size (%zu) does not fit the constant memory buffer (%zu)\n", size,
                sizeof(m_const));
        exit(-1);
    }

    void* ptr(nullptr);
    CUDA_ERR_CHECK(hipGetSymbolAddress(&ptr, HIP_SYMBOL(m_const)));
    CUDA_ERR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(m_const), data, size));
    return ptr;
}

#ifdef CUDA_PGI_WORKAROUND

struct fatCubinStruct
{
    int magic;    // Always 0x466243b1
    int version;  // Sequence number of the cubin
    void* fat;    // The pointer to the real cubin
    void* other;  // Some pointer related to the data segment
};

// Working around a SIGSEGV in __pgi_cuda_register_fat_binaryA
// occuring specifically when compiling a pybind11 moddule with nvc++.
// Perhaps, PGI folks forgot to cover this in their tricky __PGI_CUDA_LOC
// anchor logic. This code is not portable and should go away once
// there is a fix from NVIDIA.

extern "C" void* __pgi_cuda_register_fat_binaryA(fatCubinStruct* fatCubin, void** pgi_cuda_loc);

extern "C" void* __pgi_cuda_register_fat_binary(fatCubinStruct* fatCubin)
{
    fatCubinStruct* __PGI_CUDA_LOC = (fatCubinStruct*) dlsym(nullptr, "__PGI_CUDA_LOC");
    __pgi_cuda_register_fat_binaryA(fatCubin, (void**) __PGI_CUDA_LOC);

    return __PGI_CUDA_LOC->other;
}

// Further working around cudaInvalidDeviceFunction error during hipLaunchByPtr in PGI nvc++,
// when the code in compiled as a shared library.

#    define EXPECT(var, value)                                                                                         \
        expect(var, value,                                                                                             \
               (const char*) "The supported version of __cudaRegisterFunction() expects \"" #var                       \
                             "\" value to be equal to " #value "\n")

template <typename T, typename TVal>
inline void expect(const T& var, const TVal value, const char* msg)
{
    if (var != value) {
        fprintf(stderr, "%s", msg);
        exit(-1);
    }
}

hipModule_t module = nullptr;

std::map<void*, std::pair<std::string, hipFunction_t>>* deviceFuns_ = nullptr;

// Override the internal CUDA kernels registration hook, in order to make our own index of
// the available CUDA kernels. Later on, we use this index to perform our own kernel launch
// sequence, which replaces PGI's runtime library issues.
extern "C" void __cudaRegisterFunction(void** fatCubinHandle_, const char* hostFun, char* deviceFun,
                                       const char* deviceName, int thread_limit, uint3* tid, uint3* bid, dim3* bDim,
                                       dim3* gDim, int* wSize)
{
    EXPECT(thread_limit, -1);
    EXPECT(tid, nullptr);
    EXPECT(bid, nullptr);
    EXPECT(bDim, nullptr);
    EXPECT(gDim, nullptr);
    EXPECT(wSize, nullptr);

    hipError_t err;

    if (!module) {
        // Like hipInit(), but more appropriate.
        hipSetDevice(0);

        hipDevice_t dev;
        err = hipCtxGetDevice(&dev);
        if (err != hipSuccess) {
            fprintf(stderr, "Cannot get device from context, error = %d\n", err);
            exit(-1);
        }

        hipCtx_t ctx;
        err = hipDevicePrimaryCtxRetain(&ctx, dev);
        if (err != hipSuccess) {
            fprintf(stderr, "Cannot retain device primary context, error = %d\n", err);
            exit(-1);
        }

        fatCubinStruct* fatCubinHandle = reinterpret_cast<fatCubinStruct*>(*fatCubinHandle_);
        char* fatbin = reinterpret_cast<char*>(fatCubinHandle->fat);
        unsigned char magic[] = {0x50, 0xed, 0x55, 0xba, 0x01, 0x00, 0x10, 0x00};
        if (memcmp(fatbin, magic, sizeof(magic))) {
            fprintf(stderr, "Could not match fatbin magic header\n");
            exit(-1);
        }

        std::vector<char> log(65536);

        // Set jit target from context (which we got by setting the device).
        hipJitOption options[] = {hipJitOptionTarget /*hipJitOptionTargetFromContext*/, hipJitOptionErrorLogBuffer,
                                  hipJitOptionErrorLogBufferSizeBytes};
        void* values[] = {(void*) CU_TARGET_COMPUTE_70 /*NULL*/, &log[0], (void*) log.size()};

        // Load the located CUDA fatbinary image into a "module" prepared for execution.
        // This call may also choose to JIT-compile the PTX assembly, if provided within a fatbinary.
        err = hipModuleLoadDataEx(&module, &fatbin[0], sizeof(options) / sizeof(options[0]), options, values);
        if (err != hipSuccess) {
            fprintf(stderr, "Could not load CUDA module, error = %d, log = \"%s\"\n", (int) err, &log[0]);
            exit(-1);
        }
    }

    if (!deviceFuns_) {
        // Deferred map creation, to make sure we are not overrunning its C++ static
        // constructor, as __cudaRegisterFunction() calls may happen quite early at the startup.
        deviceFuns_ = new std::map<void*, std::pair<std::string, hipFunction_t>>();
    }

    // Cache the function pointer-name mapping.
    auto& deviceFuns = *deviceFuns_;
    deviceFuns[(void*) hostFun].first = deviceName;
}

#    undef EXPECT

extern "C" hipError_t __cudaPopCallConfiguration(dim3* gridDim, dim3* blockDim, size_t* sharedMem, void*);

extern "C" int __pgiLaunchKernelFromStub(void* hostFun, void** argv, int argc)
{
    // Extract the functions pointer-name mapping.
    auto& deviceFuns = *deviceFuns_;
    auto& deviceName = deviceFuns.at(hostFun).first;
    if (deviceName == "") {
        fprintf(stderr, "Could not launch unknown CUDA function %p\n", hostFun);
        exit(-1);
    }

    hipError_t err;

    // Lookup for a previously cached function.
    hipFunction_t& func = deviceFuns[(void*) hostFun].second;
    if (!func) {
        // Load function from module.
        err = hipModuleGetFunction(&func, module, deviceName.c_str());
        if (err != hipSuccess) {
            fprintf(stderr, "Could not get CUDA kernel %s, error = %d\n", deviceName.c_str(), (int) err);
            exit(-1);
        }
    }

    // Extract the latest <<<...>>> kernel launch compute grid configuration,
    // which we will use to launch the kernel manually with CUDA Driver API.
    dim3 gridDim, blockDim;
    size_t sharedMem;
    hipStream_t stream;
    __cudaPopCallConfiguration(&gridDim, &blockDim, &sharedMem, &stream);

    // Launch the kernel with CUDA Driver API.
    debug::printf("GPU kernel <(%d, %d, %d), (%d, %d, %d)>\n", (int) gridDim.x, (int) gridDim.y, (int) gridDim.z,
                  (int) blockDim.x, (int) blockDim.y, (int) blockDim.z);
    err = hipModuleLaunchKernel(func, gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z, sharedMem, stream,
                         argv, NULL);
    if (err != hipSuccess) {
        fprintf(stderr, "Could not launch CUDA kernel %s, error = %d\n", deviceName.c_str(), (int) err);
        exit(-1);
    }

    // We do not synchronize (wait) for the kernel to finish here, as it
    // should be provided by the caller. PGI, please manage at least this
    // simple task!

    return err;
}
#endif  // CUDA_PGI_WORKAROUND
