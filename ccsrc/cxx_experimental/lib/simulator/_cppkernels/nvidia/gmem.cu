//   Copyright 2021 <Huawei Technologies Co., Ltd>
//
//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at
//
//       http://www.apache.org/licenses/LICENSE-2.0
//
//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.

#include "check.h"
#include "gmem.h"

#include <cstdio>
#include <map>
#include <memory>

using namespace std;

bool GlobalMemory::init()
{
    if (fatalError != hipSuccess) {
        return false;
    }

    return true;
}

bool GlobalMemory::isAvailable()
{
    if (!init()) {
        return false;
    }

    return (ngpus > 0);
}

void* GlobalMemory::alloc(size_t size)
{
#define MALLOC_ALIGNMENT 256

    if (!init()) {
        return NULL;
    }

    if (!gmem) {
        return NULL;
    }

    if (ptr + size + MALLOC_ALIGNMENT > gmem + szgmem) {
        return NULL;
    }

    void* result = ptr;
    ptr += size;

    ptrdiff_t alignment = (ptrdiff_t) ptr % MALLOC_ALIGNMENT;
    if (alignment) {
        ptr += MALLOC_ALIGNMENT - alignment;
    }

    return result;
}

// Reset free memory pointer to the beginning of preallocated buffer.
void GlobalMemory::free()
{
    if (!init()) {
        return;
    }

    ptr = gmem;
}

// Check whether the specified memory address belongs to GPU memory allocation.
bool GlobalMemory::isAllocatedOnGPU(const void* ptr)
{
    if (!init()) {
        return false;
    }

    if (!gmem) {
        return false;
    }

    if ((ptr >= gmem) && (ptr <= gmem + szgmem)) {
        return true;
    }

    return false;
}

hipError_t GlobalMemory::set(void* dst, const int val, size_t size)
{
    if (!init()) {
        return {hipErrorNoDevice};
    }

    hipError_t hipError_t;
    CUDA_ERR_CHECK(hipError_t = hipMemset(dst, val, size));
    if (hipError_t != hipSuccess) {
        fatalError = hipError_t;
        return {fatalError};
    }

    return {hipError_t};
}

hipError_t GlobalMemory::copy(void* dst, const void* src, size_t size)
{
    if (!init()) {
        return {hipErrorNoDevice};
    }

    hipMemcpyKind kind = hipMemcpyDeviceToHost;
    if (isAllocatedOnGPU(dst) && isAllocatedOnGPU(src)) {
        kind = hipMemcpyDeviceToDevice;
    }
    else if (!isAllocatedOnGPU(dst) && !isAllocatedOnGPU(src)) {
        kind = hipMemcpyHostToHost;
    }
    else if (isAllocatedOnGPU(dst) && !isAllocatedOnGPU(src)) {
        kind = hipMemcpyHostToDevice;
    }

    hipError_t hipError_t;
    CUDA_ERR_CHECK(hipError_t = hipMemcpy(dst, src, size, kind));
    if (hipError_t != hipSuccess) {
        fatalError = hipError_t;
        return {fatalError};
    }

    return {hipError_t};
}

hipError_t GlobalMemory::getLastError()
{
    // If GPU is not initialized, then there is either no
    // device or fatal error during initialization.
    if (!init()) {
        return {fatalError};
    }

    return {hipGetLastError()};
}

GlobalMemory::GlobalMemory() : fatalError(hipSuccess), ngpus(0), gmem(NULL), ptr(NULL)
{
    hipError_t hipError_t;

#define CUDA_RETURN_ON_ERR(x)                                                                                          \
    do {                                                                                                               \
        CUDA_ERR_CHECK(x);                                                                                             \
        if (hipError_t != hipSuccess) {                                                                                \
            fatalError = hipError_t;                                                                                    \
            return;                                                                                                    \
        }                                                                                                              \
    } while (0)

    CUDA_ERR_CHECK(hipError_t = hipGetDeviceCount(&ngpus));
    if ((hipError_t != hipSuccess) && (hipError_t != hipErrorNoDevice)) {
        fatalError = hipError_t;
        return;
    }

    if (!ngpus) {
        return;
    }

    // Preallocate 85% of GPU memory to save on costly allocations/deallocations.
    size_t available, total;
    CUDA_RETURN_ON_ERR(hipError_t = hipMemGetInfo(&available, &total));

    szgmem = (size_t) (0.85 * available);

    CUDA_RETURN_ON_ERR(hipError_t = hipMalloc(&gmem, szgmem));

    ptr = gmem;

#undef CUDA_RETURN_ON_ERR
}

GlobalMemory::~GlobalMemory()
{
    hipFree(gmem);
}

// ==============================================================================

static std::map<int, std::unique_ptr<GlobalMemory>> global_memory_;

GlobalMemory* get_memory_on_gpu(int device)
{
    if (auto it = global_memory_.find(device); it != std::end(global_memory_)) {
        return it->second.get();
    }
    else {
        auto [it_new, _] = global_memory_.emplace(device, std::make_unique<GlobalMemory>());
        return it_new->second.get();
    }
}

void release_memory_on_gpu(int device)
{
    global_memory_.erase(device);
}
